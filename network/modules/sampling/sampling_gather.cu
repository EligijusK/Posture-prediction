#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <stdio.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void gatherpointLauncher(int b, int n, int m, const float *inp, const int *idx, float *out);
__global__ void gatherpointKernel(int b, int n, int m, const float *__restrict__ inp, const int *__restrict__ idx,float *__restrict__ out);

void scatteraddpointLauncher(int b, int n, int m, const float *out_g, const int *idx, float *inp_g);
__global__ void scatteraddpointKernel(int b, int n, int m, const float *__restrict__ out_g, const int *__restrict__ idx,float *__restrict__ inp_g);


__global__ void scatteraddpointKernel(int b, int n, int m, const float *__restrict__ out_g, const int *__restrict__ idx,float *__restrict__ inp_g) {
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int j = blockIdx.y * blockDim.x + threadIdx.x; j < m; j += blockDim.x * gridDim.y) {
            int a = idx[i * m + j];

            atomicAdd(&inp_g[(i * n + a) * 3 + 0], out_g[(i * m + j) * 3 + 0]);
            atomicAdd(&inp_g[(i * n + a) * 3 + 1], out_g[(i * m + j) * 3 + 1]);
            atomicAdd(&inp_g[(i * n + a) * 3 + 2], out_g[(i * m + j) * 3 + 2]);
        }
    }
}

void scatteraddpointLauncher(int b, int n, int m, const float *out_g, const int *idx, float *inp_g) {
    scatteraddpointKernel<<<dim3(2, 8, 1), 512>>>(b, n, m, out_g, idx, inp_g);
}

__global__ void gatherpointKernel(int b, int n, int m, const float *__restrict__ inp, const int *__restrict__ idx,float *__restrict__ out) {
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
      for (int j = blockIdx.y * blockDim.x + threadIdx.x; j < m; j += blockDim.x * gridDim.y) {
        int a = idx[i * m + j];

        out[(i * m + j) * 3 + 0] = inp[(i * n + a) * 3 + 0];
        out[(i * m + j) * 3 + 1] = inp[(i * n + a) * 3 + 1];
        out[(i * m + j) * 3 + 2] = inp[(i * n + a) * 3 + 2];
      }
    }
  }

void gatherpointLauncher(int b, int n, int m, const float *inp, const int *idx, float *out) {
    gatherpointKernel<<<dim3(2, 8, 1), 512>>>(b, n, m, inp, idx, out);
}

int gatherCudaForward(at::Tensor inpTensor, at::Tensor idxTensor, at::Tensor outTensor) {
    if(inpTensor.ndimension() != 3 || inpTensor.size(2) != 3) {
        printf("GatherPoint expects (batch_size, num_points, 3) inpTensor");
        return -1;
    }

    int b = inpTensor.size(0);
    int n = inpTensor.size(1);

    if(idxTensor.ndimension() != 2 || idxTensor.size(0) != b) {
        printf("GatherPoint expects (batch_size, num_result) idxTensor");
        return -1;
    }

    int m = idxTensor.size(1);
    
    const float *inp = inpTensor.flatten().data_ptr<float>();
    int *idx = idxTensor.flatten().data_ptr<int>();
    float *out = outTensor.flatten().data_ptr<float>();
    
    gatherpointLauncher(b, n, m, inp, idx, out);

    hipError_t err = hipGetLastError();
    
    if (err != hipSuccess) {
        printf("Error in nnd Output: %s\n", hipGetErrorString(err));
        return -1;
    }

    return 1;
}

int gatherCudaBackward(at::Tensor inpTensor, at::Tensor idxTensor, at::Tensor outTensor, at::Tensor outGradTensor, at::Tensor inGradTensor) {
    if(inpTensor.ndimension() != 3 || inpTensor.size(2) != 3) {
        printf("GatherPoint expects (batch_size, num_points, 3) inpTensor");
        return -1;
    }

    int b = inpTensor.size(0);
    int n = inpTensor.size(1);
    
    if(idxTensor.ndimension() != 2 || idxTensor.size(0) != b) {
        printf("GatherPoint expects (batch_size, num_result) idxTensor");
        return -1;
    }

    int m = idxTensor.size(1);

    const float *inp = inpTensor.flatten().data_ptr<float>();
    const int *idx = idxTensor.flatten().data_ptr<int>();

    if(outGradTensor.ndimension() != 3 || outGradTensor.size(0) != b || outGradTensor.size(1) != m || outGradTensor.size(2) != 3) {
        printf("GatherPointGradGpuOp expects (batch_size,num_result,3) outGradTensor");
        return -1;
    }

    const float *out_g = outGradTensor.flatten().data_ptr<float>();
    float *inp_g = inGradTensor.flatten().data_ptr<float>();

    scatteraddpointLauncher(b, n, m, out_g, idx, inp_g);

    hipError_t err = hipGetLastError();
    
    if (err != hipSuccess) {
        printf("Error in nnd Output: %s\n", hipGetErrorString(err));
        return -1;
    }

    return 1;
}