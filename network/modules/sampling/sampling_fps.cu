#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <stdio.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void farthestpointsamplingLauncher(int b, int n, int m, const float *inp, float *temp, int *out);
__global__ void farthestpointsamplingKernel(int b, int n, int m, const float *__restrict__ dataset, float *__restrict__ temp, int *__restrict__ idxs);

__global__ void farthestpointsamplingKernel(int b, int n, int m, const float *__restrict__ dataset, float *__restrict__ temp, int *__restrict__ idxs) {
    if (m <= 0) return;

    const int BlockSize = 512;
    
    __shared__ float dists[BlockSize];
    __shared__ int dists_i[BlockSize];
    
    const int BufferSize = 3072;
    __shared__ float buf[BufferSize * 3];
    
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        int old = 0;

        if (threadIdx.x == 0) idxs[i * m + 0] = old;
        
        for (int j = threadIdx.x; j < n; j += blockDim.x) {
            temp[blockIdx.x * n + j] = 1e38;
        }

        for (int j = threadIdx.x; j < min(BufferSize, n) * 3; j += blockDim.x) {
            buf[j] = dataset[i * n * 3 + j];
        }
        __syncthreads();

        for (int j = 1; j < m; j++) {
            int besti = 0;
            float best = -1;

            float x1 = dataset[i * n * 3 + old * 3 + 0];
            float y1 = dataset[i * n * 3 + old * 3 + 1];
            float z1 = dataset[i * n * 3 + old * 3 + 2];

            for (int k = threadIdx.x; k < n; k += blockDim.x) {
                float td = temp[blockIdx.x*n+k];
                float x2, y2, z2;

                if (k < BufferSize){
                    x2 = buf[k * 3 + 0];
                    y2 = buf[k * 3 + 1];
                    z2 = buf[k * 3 + 2];
                } else {
                    x2=dataset[i * n * 3 + k * 3 + 0];
                    y2=dataset[i * n * 3 + k * 3 + 1];
                    z2=dataset[i * n * 3 + k * 3 + 2];
                }
               
                float d = (x2-x1) * (x2-x1) + (y2-y1) * (y2-y1) + (z2-z1) * (z2-z1);
                float d2 = min(d, td);

                if (d2 != td) temp[blockIdx.x * n + k] = d2;
                
                if (d2 > best){
                    best = d2;
                    besti = k;
                }
            }

            dists[threadIdx.x] = best;
            dists_i[threadIdx.x] = besti;

            for (int u = 0; (1 << u)< blockDim.x; u++){
                __syncthreads();

                if (threadIdx.x<(blockDim.x >> (u + 1))) {
                    int i1 = (threadIdx.x * 2) << u;
                    int i2 = (threadIdx.x * 2 + 1) << u;

                    if (dists[i1] < dists[i2]) {
                        dists[i1] = dists[i2];
                        dists_i[i1] = dists_i[i2];
                    }
                }
            }
            __syncthreads();

            old = dists_i[0];

            if (threadIdx.x == 0) idxs[i * m + j] = old;
        }
    }
}

//require 32*n working space
void farthestpointsamplingLauncher(int b, int n, int m, const float *inp, float *temp, int *out) {
    farthestpointsamplingKernel<<<32,512>>>(b, n, m, inp, temp, out);
}

int farthestPointSamplingCuda(const int nsamples, at::Tensor inpTensor, at::Tensor outTensor) {
    if(inpTensor.ndimension() != 3 || inpTensor.size(2) != 3) {
        printf("FarthestPointSample expects (batch_size, num_points, 3) inp shape.");
        return -1;
    }
    
    int m = nsamples;
    int b = inpTensor.size(0);
    int n = inpTensor.size(1);

    const float *inp = inpTensor.flatten().data_ptr<float>();
    int *out = outTensor.flatten().data_ptr<int>();

    auto options = torch::TensorOptions()
        .device(inpTensor.device().type(), inpTensor.device().index())
        .dtype(torch::kFloat32)
        .requires_grad(false);
    at::Tensor tempTensor = torch::empty({32, n}, options);

    float *temp = tempTensor.flatten().data_ptr<float>();

    farthestpointsamplingLauncher(b, n, m, inp, temp, out);

    hipError_t err = hipGetLastError();
    
    if (err != hipSuccess) {
        printf("Error in nnd Output: %s\n", hipGetErrorString(err));
        return -1;
    }

    return 1;
}